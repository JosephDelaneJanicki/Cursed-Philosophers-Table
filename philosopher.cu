// this is the file that will make the philosopher object and map philosophers to threads. from there the gpu will handle the philosophers
// philosopher.cu
#include <hip/hip_runtime.h>


// Philosopher definition
class Philosopher {
public:
    // TODO: Define philosopher states and actions 

};

// kernel definition
__global__ void philosopherSimulation(Philosopher* philosophers, int numPhilosophers) {
    int philosopherIdx = threadIdx.x;
    
    // TODO: Simulate philosopher actions (picking up forks, eating, thinking, ect)
}

