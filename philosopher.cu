#include "hip/hip_runtime.h"
// this is the file that will make the philosopher object and map philosophers to threads. from there the gpu will handle the philosophers
// philosopher.cu
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <cstdio>

//initializa mutex
/*NOTE: avoid mutexes in philosopher class, mutexes would be best in the fork class and kernel to minimize sequential time and maximize parrallel time. philosophers do the things 
forks are the rescources so let the forks protect themselves with mutexes philosophers are supposed to be fork greedy */
__device__ int mutex; // Declare a mutex variable

// Philosopher definition
class Philosopher {
    public:

        __device__ void think() {
            int startTime = clock();
            while (clock() - startTime < 1000000) { /*the philosopher is thinking, using a busy wait loop to simulate it for now will add logic as needed later on*/ }
        }

        __device__ void tryToPickUpForks(Fork& leftFork, Fork& rightFork){

            if (leftFork.isAvailable())  {leftFork.pickUp(); think();}  // by thinking here i can garuntee deadlock since now the forks are picked up one at a time
            else  think(); tryToPickUpForks;

            while(rightFork.isAvailable() == false) think();
            rightFork.pickUp();
            eat(leftFork, rightFork);
            leftFork.putDown();
            rightFork.putDown();
        }
        /*by picking up the forks at the same time and dropping at the same time deadlock should be avoided.
        this i believe will be my first solution before going down the "cursed rought"*/
        __device__ void tryToPickUpForksAvoidDeadlock(Fork& leftFork, Fork& rightFork) {
            if (leftFork.isAvailable() && rightFork.isAvailable()) {
                leftFork.pickUp();
                rightFork.pickUp();
                eat(leftFork, rightFork);
                leftFork.putDown();
                rightFork.putDown();
            } 
            else think();
        }

        __device__ void eat(Fork& leftFork, Fork& rightFork){
            int startTime = clock();
            while (clock() - startTime < 1000000) { /*the philosopher is eating, using a busy wait loop to simulate it for now. will add logic as needed later on*/ }
        }
        __device__ void kill(Fork& leftFork, Fork& rightFork){
            dead = true;
            leftFork.putDown();
            rightFork.putDown();
        }
        //dead getter method
        __device__ bool isDead(){
            return dead;
        }

    private:
        bool dead = false;
};
// Fork object to allow solutions that involve changing something about the fork/forks. Forks are picked up and put down by philosophers
class Fork{
    private:

        bool available = true;  //fork availability to pick up, by default they are available
        int* mutex; // Private mutex for each fork

    public:

        int temprature {}; // how hot or cold the fork is to the touch

        // this method checks if a fork is available since this is a read only method i dont have a mutex here
        __device__ bool isAvailable() {
            return available;
        }

        // this method allows the fork to be picked up
        __device__ bool pickUp() {
            while (atomicExch(mutex, 1) != 0) {
            // Wait while the mutex is locked
            }
            /*other philosophers need to be able to read the fork object to check availability. 
            I also want to minimize mutext time so im closing the mutex imediately when available value changes*/ 
            if (available) {
                available = false;
                atomicExch(mutex, 0); // Release the mutex 
                return true; // Successfully picked up the fork
            }       
            else {
                atomicExch(mutex, 0); // Release the mutex
                return false; // Fork is not available
            }
        }

        // I opted not to use a mutex on this function since only one philosopher will be attempting to use this method per fork at any time
        __device__ void putDown() {
            available = true; // Fork is now available for other philosophers
        }
};

// kernel definitions
// in a way we can these about this as the table the philosophers eat at

// philosophers as blocks kernel simpler but may be less GPU efficient
__global__ void philosophersAsBlocks(Philosopher* philosophers, Fork* forks, int numPhilosophers) {
    //thread mapping
    int philosopherIdx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate a unique index for each philosopher
    int leftForkIdx = philosopherIdx; // Fork on the left side of the philosopher
    int rightForkIdx = (philosopherIdx + 1) % numPhilosophers; // Fork on the right side of the philosopher
    
}

// philosophers and forks as threads will require more synchronization but is more GPU efficient
__global__ void philosophersAsThreads(Philosopher* philosophers, Fork* forks, int numPhilosophers,int iterations) {
    //thread mapping
    int philosopherIdx = threadIdx.x; // one thread per philosopher
    int leftForkIdx = philosopherIdx; // Fork on the left side of the philosopher
    int rightForkIdx = (philosopherIdx + 1) % numPhilosophers; // Fork on the right side of the philosopher

    for (int iteration = 0; iteration < iterations; ++iteration) {
        // Philosopher actions
        int philosopherId = philosopherIdx; // Get the philosopher's ID
        Philosopher& philosopher = philosophers[philosopherId];

        int randomChoice = rand() % 2; // generate random number for the thread to choose a random solution if any

        //check if the philosopher is dead and skip the iteration if they are
        bool isDead = philosophers[philosopherIdx].isDead();
        if(isDead) continue;

        // switch case that handels the choice of solution. as i make more solutions i can implement them in a case here and increase the int pool in randomChoice accordingly
        switch (randomChoice) {
            case 0:
                //first case allows deadlock to happen eat method is called in try to pick up forks
                printf("Philosopher %d is thinking.\n", philosopherId);
                philosophers[philosopherId].think();
                //this case is ordinary and avoids deadlock by having the philosopher pick up both forks at the same time
                printf("Philosopher %d is trying to pick up forks.\n", philosopherId);
                philosophers[philosopherId].tryToPickUpForks(forks[leftForkIdx],forks[rightForkIdx]);
                break;
            case 1:
                printf("Philosopher %d is thinking.\n", philosopherId);
                philosophers[philosopherId].think();
                printf("Philosopher %d is trying to pick up forks.\n", philosopherId);
                philosophers[philosopherId].tryToPickUpForksAvoidDeadlock(forks[leftForkIdx],forks[rightForkIdx]);
                break;
    }
        philosophers[philosopherId].think();
        philosophers[philosopherId].tryToPickUpForks(forks[leftForkIdx],forks[rightForkIdx]);

        // Synchronize threads to avoid conflicts
        __syncthreads();
    }
}

// this is a testing kernal I will call apon when debugging.
// the idea is this is a single solution table of 5 philosophers I can use to test and garuntee any one solution no matter how absurd does infact, end deadlock
// I'll base it on the philosophers as threads kernel
__global__ void testDeadlockResolution(Philosopher* philosophers, Fork* forks, int numPhilosophers = 5){
int philosopherIdx = threadIdx.x;
int leftForkIdx = philosopherIdx;
int rightForkIdx = (philosopherIdx + 1) % numPhilosophers;
}