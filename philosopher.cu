// this is the file that will make the philosopher object and map philosophers to threads. from there the gpu will handle the philosophers
// philosopher.cu
#include <hip/hip_runtime.h>


// Philosopher definition
class Philosopher {
public:

    // TODO: Define philosopher states and actions 

    //Due to Philosopher being an object its best to handle device memory with device functions made below
    void allocateToDevice(Philosopher** devicePtr){

        hipMalloc((void**)devicePtr,sizeof(Philosopher));

        hipMemcpy(*devicePtr, this, sizeof(Philosopher), hipMemcpyHostToDevice);

    }
    void freeDeviceMemory(){
        hipFree(this);
    }
};

class Fork{
public:
    
};

// kernel definition
//in a way we can think about this as the table itself
__global__ void philosopherSimulation(Philosopher* philosophers, int numPhilosophers) {
    //maps each philosopher to one thread
    int philosopherIdx = threadIdx.x;
    
    // TODO: Simulate philosopher actions (picking up forks, eating, thinking, ect)
}

// this is a testing kernal I will call apon when debugging.
// the idea is this is a single solution table of 5 philosophers I can use to test and garuntee any one solution no matter how absurd does infact, end deadlock
__global__ void testDeadlockResolution(Philosopher* philosophers, Fork* forks, int numPhilosophers){
    int philosopherIdx = threadIdx.x;
    
}

