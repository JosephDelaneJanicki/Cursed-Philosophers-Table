#include "hip/hip_runtime.h"
// this is the file that will make the philosopher object and map philosophers to threads. from there the gpu will handle the philosophers
// philosopher.cu
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

// Philosopher definition
class Philosopher {
public:

    // TODO: Define philosopher states and actions 

    /*NOTE: the busy wait loops are temporary measures to provide something the kernel can use to 
    simulate the philosphers they consume CPU cycles and generally arent cuda friendly. once i have the philosophers in deadlock
    ill go back and replace these with synchronization primitives*/
    __device__ void think() {

        int startTime = clock();
        while (clock() - startTime < 1000000) { /*the philosopher is thinking, using a busy wait loop to simulate it for now will add logic as needed later on*/ }
    }
    __device__ void tryToPickUpForks(Fork& leftFork, Fork& rightFork){
        if (leftFork.available)  {leftFork.pickUp(); think();}  // by thinking here i can garuntee deadlock since now the forks are picked up one at a time
        else  think(); tryToPickUpForks;
        while(rightFork.isAvailable() == false) think();
        rightFork.pickUp();
        eat(leftFork, rightFork);
        leftFork.putDown();
        rightFork.putDown();
    }
    /*by picking up the forks at the same time and dropping at the same time deadlock should be avoided.
    this i believe will be my first solution before going down the "cursed rought"*/
    __device__ void tryToPickUpForksAvoidDeadlock(Fork& leftFork, Fork& rightFork) {
    if (leftFork.isAvailable() && rightFork.isAvailable()) {
        leftFork.pickUp();
        rightFork.pickUp();
        eat(leftFork, rightFork);
        leftFork.putDown();
        rightFork.putDown();
    } 
    else think();
}
    __device__ void eat(Fork& leftFork, Fork& rightFork){

        int startTime = clock();
        while (clock() - startTime < 1000000) { /*the philosopher is eating, using a busy wait loop to simulate it for now. will add logic as needed later on*/ }

    }

    //Due to Philosopher being an object its best to handle device memory with device functions made below
    void allocateToDevice(Philosopher** devicePtr){

        hipMalloc((void**)devicePtr,sizeof(Philosopher));

        hipMemcpy(*devicePtr, this, sizeof(Philosopher), hipMemcpyHostToDevice);

    }
    void freeDeviceMemory(){
        hipFree(this);
    }
};
// Fork object to allow solutions that involve changing something about the fork/forks. Forks are picked up and put down by philosophers
class Fork{
public:
    int temprature {}; // how hot or cold the fork is to the touch
    bool available = true;  //fork availability to pick up, by default they are available

    // this method 
    bool isAvailable() {
        // You can use atomic operations or locks here for thread safety
        // For example, you can use atomic operations like atomicCAS in CUDA
        return available;
    }
    // this method allows the fork to be picked up
    bool pickUp(){
        
        if (available) available = false;
        else return false;

    }

    // this method allows the fork to be put down
    void putDown() {available = true;}
};

// kernel definition
//in a way we can think about this as the table itself
__global__ void philosopherSimulation(Philosopher* philosophers, Fork* forks, int numPhilosophers) {
    //thread mapping
    int philosopherIdx = blockIdx.x; // one block per philosopher
    int forkIdx = threadIdx.x; // one thread per fork
    

    // TODO: Simulate philosopher actions (picking up forks, eating, thinking, ect)
}

// this is a testing kernal I will call apon when debugging.
// the idea is this is a single solution table of 5 philosophers I can use to test and garuntee any one solution no matter how absurd does infact, end deadlock
__global__ void testDeadlockResolution(Philosopher* philosophers, Fork* forks, int numPhilosophers = 5){
    int philosopherIdx = blockIdx.x; 
    int forkIdx = threadIdx.x;
    
    
}

