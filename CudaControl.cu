#include "hip/hip_runtime.h"
//this file is to test that i can compile CUDA and cpp 20. the idea much like with control is that this file is super simple and 
// for sure has no logical issue. any error MUST be with how compilation is set up
#include <iostream>
#include <format>

// Cuda kernel for GPU
__global__ void cudaKernel() {
    const char s[] = "Hello from GPU!";
    printf("%s\n", s);
}

int main() {
    // CPU Code
    const char s[] = "Hello from CPU!";
    std::cout << std::format("Host: {}\n", s);

    // Launch the CUDA kernel
    cudaKernel<<<1, 1>>>();

    // Wait for GPU to finish
    hipDeviceSynchronize();

    return 0;
}
